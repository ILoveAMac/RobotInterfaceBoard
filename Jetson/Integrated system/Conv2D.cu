#include "hip/hip_runtime.h"
//
// Created by Wihan on 2024-09-08.
//

#include "Conv2D.cuh"
#include <hip/hip_runtime.h>
#include <cmath>

// CUDA kernel for Conv2D forward pass with batch normalization and ReLU
__global__ void conv2dForwardKernel(const float *input, float *output, const float *weights, const float *gamma,
                                    const float *beta, const float *runningMean, const float *runningVar,
                                    const int inputHeight, const int inputWidth, const int inputChannels,
                                    const int kernelSize,
                                    const int stride, const int padding, const int outputHeight,
                                    const int outputWidth)
{
    // Get the index of the thread in the output tensor
    const int filter = blockIdx.z;                       // Channel index
    const int h = blockIdx.y * blockDim.y + threadIdx.y; // Output height index
    const int w = blockIdx.x * blockDim.x + threadIdx.x; // Output width index

    // Where are we now:
    // We are a specific filter given by the filter index
    // On the input image we are at position (h, w).
    // (h, w) is the top left corner of the area where we will preform convolution
    // Now, we must preform the convolution operation at position (h, w) using the specific filter
    // The size of the filter is given by kernelSize, and we use it to ensure we preform the convolution within
    // the bounds of the kernel

    // Additional things to take not of:
    // We are using padding. The padding will be used to keep the input the same size as the output.
    // We are using a stride. The stride will affect the output size

    // Check that we are within bounds of the output tensor
    if (h < outputHeight && w < outputWidth)
    {
        // Accumulator for result
        float sum = 0.0f;

        // For every input channel
        for (int inputChannel = 0; inputChannel < inputChannels; inputChannel++)
        {
            // Two for loops to move over the kernel, the kernel is always square
            for (int kh = 0; kh < kernelSize; kh++)
            {
                for (int kw = 0; kw < kernelSize; kw++)
                {
                    // Calculate our position on the input tensor
                    // kh and kw are the row and column offsets within the kernel
                    const int inputH = h * stride - padding + kh;
                    const int inputW = w * stride - padding + kw;

                    if (inputH >= 0 && inputH < inputHeight && inputW >= 0 && inputW < inputWidth)
                    {
                        // Calculate the index in the flattened weights array
                        const int weightIndex = ((filter * inputChannels + inputChannel) * kernelSize * kernelSize) + (kh * kernelSize + kw);
                        sum += input[(inputChannel * inputHeight + inputH) * inputWidth + inputW] * weights[weightIndex];
                    }
                    // else the bounds are invalid,
                    // thus we treat this as padding with zeros and do not add anything to the sum
                }
            }
        }

        // Apply batch normalization
        const float mean = runningMean[filter];
        const float var = runningVar[filter];
        const float gammaVal = gamma[filter];
        const float betaVal = beta[filter];
        sum = gammaVal * (sum - mean) / sqrtf(var + 1e-8) + betaVal;

        // Apply LeRelu
        sum = (sum > 0.0f) ? sum : 0.1f * sum;

        output[(filter * outputHeight + h) * outputWidth + w] = sum;
    }
}

// CUDA kernel for Conv2D forward pass with batch normalization and Leaky ReLU using shared memory
__global__ void conv2dForwardKernelShared(const float *input, float *output, const float *weights, const float *gamma,
                                          const float *beta, const float *runningMean, const float *runningVar,
                                          const int inputHeight, const int inputWidth, const int inputChannels,
                                          const int kernelSize,
                                          const int stride, const int padding, const int outputHeight,
                                          const int outputWidth)
{
    // Calculate output coordinates
    const int filter = blockIdx.z;                           // Output channel (filter) index
    const int h_out = blockIdx.y * blockDim.y + threadIdx.y; // Output height index
    const int w_out = blockIdx.x * blockDim.x + threadIdx.x; // Output width index

    // Check if within output bounds
    if (h_out >= outputHeight || w_out >= outputWidth)
        return;

    // Calculate the corresponding input coordinates
    const int h_in = h_out * stride - padding;
    const int w_in = w_out * stride - padding;

    // Shared memory for input tile
    extern __shared__ float sharedInput[];
    // Size of shared memory per input channel
    const int sharedMemPerChannel = (blockDim.y + kernelSize - 1) * (blockDim.x + kernelSize - 1);

    // Load input tile into shared memory for each input channel
    for (int c = 0; c < inputChannels; ++c)
    {
        for (int i = threadIdx.y; i < blockDim.y + kernelSize - 1; i += blockDim.y)
        {
            for (int j = threadIdx.x; j < blockDim.x + kernelSize - 1; j += blockDim.x)
            {
                int h = h_in + i;
                int w = w_in + j;
                float value = 0.0f;
                if (h >= 0 && h < inputHeight && w >= 0 && w < inputWidth)
                {
                    int inputIdx = (c * inputHeight + h) * inputWidth + w;
                    value = input[inputIdx];
                }
                int sharedIdx = c * sharedMemPerChannel + i * (blockDim.x + kernelSize - 1) + j;
                sharedInput[sharedIdx] = value;
            }
        }
    }

    // Synchronize to ensure all data is loaded into shared memory
    __syncthreads();

    // Perform convolution
    float sum = 0.0f;
    for (int c = 0; c < inputChannels; ++c)
    {
        for (int kh = 0; kh < kernelSize; ++kh)
        {
            for (int kw = 0; kw < kernelSize; ++kw)
            {
                int sharedIdx = c * sharedMemPerChannel + (threadIdx.y + kh) * (blockDim.x + kernelSize - 1) + (threadIdx.x + kw);
                int weightIdx = ((filter * inputChannels + c) * kernelSize * kernelSize) + (kh * kernelSize + kw);
                sum += sharedInput[sharedIdx] * weights[weightIdx];
            }
        }
    }

    // Apply batch normalization
    const float mean = runningMean[filter];
    const float var = runningVar[filter];
    const float gammaVal = gamma[filter];
    const float betaVal = beta[filter];
    sum = gammaVal * (sum - mean) / sqrtf(var + 1e-8f) + betaVal;

    // Apply Leaky ReLU
    sum = (sum > 0.0f) ? sum : 0.1f * sum;

    // Write the result to the output tensor
    int outputIdx = (filter * outputHeight + h_out) * outputWidth + w_out;
    output[outputIdx] = sum;
}

Conv2D::Conv2D(const int kernelSize, const int numFilters, const int stride, const int padding,
               const std::string &layerName, const ModelLoadingHelper &ml, const int outHeight, const int outWidth,
               const int outChannels, const int inputHeight, const int inputWidth,
               const int inputChannels) : d_weights(nullptr),
                                          d_gamma(nullptr),
                                          d_beta(nullptr),
                                          d_runningMean(nullptr),
                                          d_runningVar(nullptr), ml(ml)
{
    this->kernelSize = kernelSize;
    this->numFilters = numFilters;
    this->stride = stride;
    this->padding = padding;
    this->layerName = layerName;

    // Output shape
    this->outputChannels = outChannels;
    this->outputHeight = outHeight;
    this->outputWidth = outWidth;

    // Input shape
    this->inputHeight = inputHeight;
    this->inputWidth = inputWidth;
    this->inputChannels = inputChannels;

    // Allocate scratch space for the output tensor
    // hipMallocManaged(&d_intermediate, outputHeight * outputWidth * numFilters * sizeof(float));
    hipMalloc(&d_intermediate, outputHeight * outputWidth * numFilters * sizeof(float));
}

Conv2D::~Conv2D()
{
    // Check if the pointers are not null and free GPU memory
    if (d_weights != nullptr)
    {
        hipFree(d_weights);
        d_weights = nullptr; // Set the pointer to nullptr after freeing
    }

    if (d_gamma != nullptr)
    {
        hipFree(d_gamma);
        d_gamma = nullptr;
    }

    if (d_beta != nullptr)
    {
        hipFree(d_beta);
        d_beta = nullptr;
    }

    if (d_runningMean != nullptr)
    {
        hipFree(d_runningMean);
        d_runningMean = nullptr;
    }

    if (d_runningVar != nullptr)
    {
        hipFree(d_runningVar);
        d_runningVar = nullptr;
    }
}

// This function will load all data associated with the conv layer into vectors
// based on the name of the vector and then flatten the vectors and move them to the GPU
void Conv2D::loadData()
{
    // Load the weights into a vector from a file using the ModelLoadingHelper
    const std::string convWeightPath = "darknet." + this->layerName + ".conv.weight.bin";
    const auto weights = ml.loadConv4D(convWeightPath);

    // Load the batch norm weights
    const std::string batchNormWeights = "darknet." + this->layerName + ".batchnorm.weight.bin";
    const auto bnWeights = ml.load1D(batchNormWeights);

    // Load the batch norm bias
    const std::string batchNormBias = "darknet." + this->layerName + ".batchnorm.bias.bin";
    const auto bnBias = ml.load1D(batchNormBias);

    // Load the batch norm running mean
    const std::string batchRunningMean = "darknet." + this->layerName + ".batchnorm.running_mean.bin";
    const auto bnRunningMean = ml.load1D(batchRunningMean);

    // Load the batch norm running var
    const std::string batchRunningVar = "darknet." + this->layerName + ".batchnorm.running_var.bin";
    const auto bnRunningVar = ml.load1D(batchRunningVar);

    // Now vectors must be flattened and assigned to the GPU

    // Weights
    const auto f_weights = flatten4D(weights);
    allocateAndCopyUnifiedMemory(f_weights, d_weights);

    // bnWeights
    allocateAndCopyUnifiedMemory(bnWeights, d_gamma);

    // bnBias
    allocateAndCopyUnifiedMemory(bnBias, d_beta);

    // bnRunningMean
    allocateAndCopyUnifiedMemory(bnRunningMean, d_runningMean);

    // bnRunningVar
    allocateAndCopyUnifiedMemory(bnRunningVar, d_runningVar);
}

float *Conv2D::forward(const float *input)
{
    // Block and grid sizes to launch the CUDA Kernel
    dim3 blockDim(16, 16); // 16x16 threads per block (256 which is dividable by 32 as warps run in groups of 32)
    // Calculation of the grid dimensions below ensures that we always have enough blocks to cover the whole image
    dim3 gridDim((outputWidth + blockDim.x - 1) / blockDim.x, (outputHeight + blockDim.y - 1) / blockDim.y, numFilters);

    // Launch the CUDA Kernel
    // conv2dForwardKernel<<<gridDim, blockDim>>>(input, d_intermediate, d_weights, d_gamma, d_beta, d_runningMean, d_runningVar,
    //                                           inputHeight, inputWidth, inputChannels, kernelSize, stride, padding,
    //                                           outputHeight, outputWidth);

    // Calculate shared memory size
    int sharedInputSizePerChannel = (blockDim.y + kernelSize - 1) * (blockDim.x + kernelSize - 1);
    size_t sharedMemorySize = inputChannels * sharedInputSizePerChannel * sizeof(float);

    // Launch the kernel
    conv2dForwardKernelShared<<<gridDim, blockDim, sharedMemorySize>>>(
        input, d_intermediate, d_weights, d_gamma, d_beta, d_runningMean, d_runningVar,
        inputHeight, inputWidth, inputChannels, kernelSize, stride, padding,
        outputHeight, outputWidth);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    return d_intermediate;
}

int Conv2D::getOutputHeight() const
{
    return this->outputHeight;
}

int Conv2D::getOutputWidth() const
{
    return this->outputWidth;
}

int Conv2D::getOutputChannels() const
{
    return this->outputChannels;
}

std::vector<float> Conv2D::flatten2D(const std::vector<std::vector<float>> &input)
{
    std::vector<float> flattened;
    for (const auto &row : input)
    {
        flattened.insert(flattened.end(), row.begin(), row.end());
    }
    return flattened;
}

std::vector<float> Conv2D::flatten4D(const std::vector<std::vector<std::vector<std::vector<float>>>> &input)
{
    std::vector<float> flattened;
    for (const auto &tensor : input)
    {
        for (const auto &matrix : tensor)
        {
            for (const auto &row : matrix)
            {
                flattened.insert(flattened.end(), row.begin(), row.end());
            }
        }
    }
    return flattened;
}

void Conv2D::allocateAndCopyUnifiedMemory(const std::vector<float> &flattenedData, float *&d_ptr)
{
    // Calculate the size of the flattened data in bytes
    const size_t dataSize = flattenedData.size() * sizeof(float);

    // Step 1: Allocate GPU memory
    const hipError_t err = hipMalloc(&d_ptr, dataSize); // Allocate memory on the GPU (device)
    if (!checkCudaError(err, "Failed to allocate GPU memory"))
    {
        return; // Handle error: early exit if allocation failed
    }

    // Step 2: Copy data from CPU to GPU
    const hipError_t memcpyErr = hipMemcpy(d_ptr, flattenedData.data(), dataSize, hipMemcpyHostToDevice);
    if (!checkCudaError(memcpyErr, "Failed to copy data to GPU memory"))
    {
        hipFree(d_ptr); // Free GPU memory if copying fails
        return;          // Handle error: early exit if memcpy failed
    }
    // d_ptr now points to the GPU memory containing the flattened data
}

bool Conv2D::checkCudaError(const hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        return false; // Return false to indicate failure
    }
    return true; // Return true if no error
}
