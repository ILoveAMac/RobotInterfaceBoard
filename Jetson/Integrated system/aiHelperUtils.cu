#include "hip/hip_runtime.h"
#include "aiHelperUtils.cuh"

aiHelperUtils::aiHelperUtils() {}

aiHelperUtils::~aiHelperUtils() {}

std::vector<std::vector<__half>> aiHelperUtils::getFinalBoundingBoxes(const __half *detections)
{
    // Vecttor to store the bounding boxes
    std::vector<std::vector<__half>> boxes;
    for (int i = 0; i < GRID_SIZE; i++)
    {
        for (int j = 0; j < GRID_SIZE; j++)
        {
            for (int b = 0; b < NUM_BOXES; b++)
            { // Loop over each bounding box in the cell
                // Calculate the starting index for the current bounding box (5 values per bounding box)
                const int index = (i * GRID_SIZE + j) * (5 * NUM_BOXES) + b * 5;

                // Extract all of the bounding boxes and store them in the boxes vector
                __half x_offset = detections[index];                                // x relative to the grid cell
                __half y_offset = detections[index + 1];                            // y relative to the grid cell
                __half w = __hmul(detections[index + 2], __float2half(IMG_HEIGHT)); // Width relative to image size
                __half h = __hmul(detections[index + 3], __float2half(IMG_WIDTH));  // Height relative to image size
                __half c = detections[index + 4];                                   // Confidence for the bounding box

                __half x_center = __hmul(__hadd(__float2half(j), x_offset), __float2half(IMG_HEIGHT / GRID_SIZE)); // Absolute x-center
                __half y_center = __hmul(__hadd(__float2half(i), y_offset), __float2half(IMG_WIDTH / GRID_SIZE));  // Absolute y-center

                std::vector<__half> box = {x_center, y_center, w, h, c};

                // Push the box to the boxes vector
                boxes.push_back(box);
            }
        }
    }

    // Perform non-maximum suppression to remove overlapping bounding boxes
    return aiHelperUtils::nonMaxSuppression(boxes);
}

cv::Mat aiHelperUtils::drawBoundingBoxes(cv::Mat frame, std::vector<std::vector<__half>> boxes)
{
    for (std::vector<__half> box : boxes)
    {
        // Extract the bounding box data
        __half x_center = box[0];
        __half y_center = box[1];
        __half w = box[2];
        __half h = box[3];
        __half c = box[4];

        // Calculate the top-left and bottom-right points of the bounding box
        __half half_two = __float2half(2.0);
        int x1 = static_cast<int>(__hsub(x_center, __hdiv(w, half_two)));
        int y1 = static_cast<int>(__hsub(y_center, __hdiv(h, half_two)));
        int x2 = static_cast<int>(__hadd(x_center, __hdiv(w, half_two)));
        int y2 = static_cast<int>(__hadd(y_center, __hdiv(h, half_two)));

        // Draw the bounding box on the image
        cv::rectangle(frame, cv::Point(x1, y1), cv::Point(x2, y2), cv::Scalar(0, 255, 0), 2);

        // Convert confidence to string and format it
        std::string label = cv::format("%.2f", c);

        // Set the position for the confidence label (above the top-left corner of the bounding box)
        int baseline = 0;
        cv::Size label_size = cv::getTextSize(label, cv::FONT_HERSHEY_SIMPLEX, 0.5, 1, &baseline);
        int label_x = std::max(x1, 0);                     // Ensure the label is inside the image boundaries
        int label_y = std::max(y1 - label_size.height, 0); // Display above the box

        // Draw the label background rectangle
        cv::rectangle(frame, cv::Point(label_x, label_y), cv::Point(label_x + label_size.width, label_y + label_size.height + baseline),
                      cv::Scalar(0, 255, 0), cv::FILLED);

        // Put the confidence text on the image
        cv::putText(frame, label, cv::Point(label_x, label_y + label_size.height),
                    cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 0, 0), 1);
    }

    return frame;
}

std::vector<std::vector<__half>> aiHelperUtils::nonMaxSuppression(std::vector<std::vector<__half>> boxes)
{
    // Remove any predictions from the list that have a confidence score less than the threshold
    // confidences are stored in the last element of the vector
    std::vector<std::vector<__half>> filtered_boxes;
    for (int i = 0; i < boxes.size(); i++)
    {
        if (__half2float(boxes[i][4]) > CONF_THRESH)
        {
            filtered_boxes.push_back(boxes[i]);
        }
    }

    // Sort the boxes based on their confidence scores, highest first
    std::sort(filtered_boxes.begin(), filtered_boxes.end(), [](const std::vector<__half> &a, const std::vector<__half> &b)
              { return a[4] > b[4]; });

    // Perform non-maximum suppression
    std::vector<std::vector<__half>> final_boxes;
    while (filtered_boxes.size() > 0)
    {
        std::vector<__half> chosenBox = filtered_boxes[0];
        // Add the chosen box to the final list
        final_boxes.push_back(chosenBox);
        // Remove the chosen box from the list
        filtered_boxes.erase(filtered_boxes.begin());

        // Remove any boxes that have a high IoU with the chosen box
        filtered_boxes.erase(std::remove_if(filtered_boxes.begin(), filtered_boxes.end(),
                                            [chosenBox](const std::vector<__half> &box)
                                            {
                                                return aiHelperUtils::iou(chosenBox, box) > IOU_NMS_THRESH;
                                            }),
                             filtered_boxes.end());
    }

    return final_boxes;
}

__half aiHelperUtils::iou(std::vector<__half> box1, std::vector<__half> box2)
{
    // Calculate the intersection area of the two boxes
    __half half_two = __float2half(2.0);
    __half box1_x1 = __hsub(box1[0], __hdiv(box1[2], half_two));
    __half box1_x2 = __hadd(box1[0], __hdiv(box1[2], half_two));
    __half box1_y1 = __hsub(box1[1], __hdiv(box1[3], half_two));
    __half box1_y2 = __hadd(box1[1], __hdiv(box1[3], half_two));

    __half box2_x1 = __hsub(box2[0], __hdiv(box2[2], half_two));
    __half box2_x2 = __hadd(box2[0], __hdiv(box2[2], half_two));
    __half box2_y1 = __hsub(box2[1], __hdiv(box2[3], half_two));
    __half box2_y2 = __hadd(box2[1], __hdiv(box2[3], half_two));

    __half x1 = hmax(box1_x1, box2_x1);
    __half y1 = hmax(box1_y1, box2_y1);
    __half x2 = hmin(box1_x2, box2_x2);
    __half y2 = hmin(box1_y2, box2_y2);

    __half zero = __float2half(0.0f);
    __half intersection = __hmul(__hmax(zero, __hsub(x2, x1)), __hmax(zero, __hsub(y2, y1)));

    __half box1_area = __hmul(fabs(box1[2]), fabs(box1[3]));
    __half box2_area = __hmul(fabs(box2[2]), fabs(box2[3]));

    return __hdiv(intersection, __hsub(__hadd(box1_area, box2_area), __hadd(intersection, __float2half(1E-6))));
}
