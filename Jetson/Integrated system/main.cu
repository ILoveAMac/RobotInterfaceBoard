// Standard Library Headers
#include <chrono>  // For timing
#include <cmath>   // For math functions
#include <cstdio>  // For printf
#include <cstring> // For memcpy
#include <iostream>
#include <string>
#include <thread>
#include <vector>

// OpenCV Headers
#include <opencv2/opencv.hpp>

// Boost Headers
#include <boost/asio.hpp>

// Project-Specific Headers
#include "Conv2D.cuh"
#include "FullyConnected.cuh"
#include "Layer.cuh"
#include "MaxPool2D.cuh"
#include "ModelLoadingHelper.h"
#include "aiHelperUtils.h"
#include "positionController.h"
#include "serialHelper.h"
#include "yolo.cuh"

using std::chrono::duration;
using std::chrono::duration_cast;
using std::chrono::high_resolution_clock;
using std::chrono::milliseconds;

int main()
{

    serialHelper serial("/dev/ttyUSB0", 9600);
    // Create a position controller
    positionController controller(0.5, 0.5, 0.1, 0.05);
    // Set the goal position
    controller.setGoal(0, 0.5, 0);
    // reset the position of the robot
    serial.resetPosition();

    // Initialize VideoCapture with default camera (index 0)
    cv::VideoCapture cap(0);
    cap.set(cv::CAP_PROP_FRAME_WIDTH, 640);
    cap.set(cv::CAP_PROP_FRAME_HEIGHT, 480);
    cap.set(cv::CAP_PROP_FPS, 30);

    // Check if the webcam opened successfully
    if (!cap.isOpened())
    {
        std::cerr << "Error: Could not open the webcam" << std::endl;
        return -1;
    }

    // Pre-allocate variables
    cv::Mat frame;
    cv::Mat resized_frame;
    std::vector<cv::Mat> channels(3);

    // Allocate device memory for the input image (3 channels, 448x448)
    float *input_image;
    hipMalloc(&input_image, 3 * 448 * 448 * sizeof(float));

    // Allocate host memory for the input image (use standard malloc or new)
    auto host_image = static_cast<float *>(malloc(3 * 448 * 448 * sizeof(float)));

    // Create a window to display the results
    cv::namedWindow("Detection", cv::WINDOW_AUTOSIZE);

    int image_counter = 0;

    aiHelperUtils aiHelper;

    yolo yolo("/home/wihan/model/");

    // Main loop
    while (true)
    {
        auto t1 = high_resolution_clock::now();
        // Capture a frame from the webcam
        cap >> frame;

        // Check if the frame is empty
        if (frame.empty())
        {
            std::cerr << "Error: Captured empty frame" << std::endl;
            break;
        }

        // Resize the image to 448x448 (input size for YOLOv1)
        cv::resize(frame, resized_frame, cv::Size(448, 448));

        // Convert the image from BGR to RGB
        cv::cvtColor(resized_frame, resized_frame, cv::COLOR_BGR2RGB);

        // Convert image to float and normalize
        resized_frame.convertTo(resized_frame, CV_32F, 1.0 / 255.0);

        // Split channels
        cv::split(resized_frame, channels);

        // Copy the data from the OpenCV Mat to the host memory (channels first format)
        for (int c = 0; c < 3; ++c)
        {
            for (int h = 0; h < 448; ++h)
            {
                for (int w = 0; w < 448; ++w)
                {
                    host_image[c * 448 * 448 + h * 448 + w] = channels[c].at<float>(h, w);
                }
            }
        }

        // Transfer the data from host memory to the GPU memory (device)
        hipMemcpy(input_image, host_image, 3 * 448 * 448 * sizeof(float), hipMemcpyHostToDevice);

        // If the current state is not a rotation, use the YOLO model to detect objects
        if (controller.getState() != State::ROTATE_TO_GOAL && controller.getState() != State::ROTATE_TO_POSITION)
        {
            // Get the bounding boxes
            std::vector<std::vector<float>> bboxes = yolo.getBoxPredictions(input_image);

            // Draw the bounding boxes
            cv::cvtColor(resized_frame, resized_frame, cv::COLOR_RGB2BGR);
            resized_frame = aiHelper.drawBoundingBoxes(resized_frame, bboxes);
        }
        else
        {
            // small delay to allow the robot to rotate
            std::this_thread::sleep_for(std::chrono::milliseconds(50));
        }

        // Display the image
        cv::imshow("Detection", resized_frame);

        // Exit if 'q' is pressed
        if (cv::waitKey(1) == 'c')
        {
            // Save the current image
            std::string filename = "img/captured_image_" + std::to_string(image_counter) + ".png";
            if (cv::imwrite(filename, 255 * resized_frame))
            {
                std::cout << "Image saved: " << filename << std::endl;
                image_counter++;
            }
            else
            {
                std::cerr << "Error: Could not save image" << std::endl;
            }
        }
        auto t2 = high_resolution_clock::now();
        duration<double, std::milli> ms_double = t2 - t1;
        std::cout << ms_double.count() << "ms\n";

        std::vector<float> position = serial.receivePosition();
        std::vector<float> velocities = controller.updateVelocities(position[0], position[1], position[2]);
        serial.sendSpeeds(velocities[1], velocities[1], velocities[0], velocities[0]);

        // print the current position
        std::cout << "Current position: " << position[0] << " " << position[1] << " " << position[2] << std::endl;

        float distance = serial.receiveDistanceSensorMeasurement(SENSE_1);
        // std::cout << "Distance sensor: " << distance << std::endl;
        if (distance < 0.2 && distance != -1)
        {
            // set velocities to 0
            serial.sendSpeeds(0, 0, 0, 0);
            break;
        }
    }

    // Release resources
    cap.release();
    cv::destroyAllWindows();

    // Free allocated memory
    hipFree(input_image);

    return 0;
}