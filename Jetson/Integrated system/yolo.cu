#include "hip/hip_runtime.h"
#include "yolo.cuh"

// Function to convert 2D vector of __half to 2D vector of float
std::vector<std::vector<float>> convertHalfToFloat(const std::vector<std::vector<__half>> &halfVec)
{
    // Determine the size of the outer vector
    size_t numRows = halfVec.size();

    // Initialize the output 2D vector with the same number of rows
    std::vector<std::vector<float>> floatVec;
    floatVec.reserve(numRows); // Reserve space to avoid multiple reallocations

    // Iterate over each row in the input 2D vector
    for (const auto &row : halfVec)
    {
        // Determine the size of the current row
        size_t numCols = row.size();

        // Initialize the inner vector for floats with the same number of columns
        std::vector<float> floatRow;
        floatRow.reserve(numCols); // Reserve space for efficiency

        // Iterate over each __half element in the current row
        for (const auto &h : row)
        {
            // Convert __half to float and add to the floatRow
            floatRow.emplace_back(__half2float(h));
        }

        // Add the converted row to the floatVec
        floatVec.emplace_back(std::move(floatRow));
    }

    return floatVec;
}

yolo::yolo(const std::string &modelPath) : MLH(ModelLoadingHelper(modelPath))
{
    // Build the model
    this->model.push_back(new Conv2D(5, 32, 2, 2, "0", MLH, 224, 224, 32, 448, 448, 3));
    this->model.push_back(new MaxPool2D(224, 224, 32, 112, 112, 32));
    this->model.push_back(new Conv2D(3, 64, 1, 1, "2", MLH, 112, 112, 64, 112, 112, 32));
    this->model.push_back(new MaxPool2D(112, 112, 64, 56, 56, 64));
    this->model.push_back(new Conv2D(1, 64, 1, 0, "4", MLH, 56, 56, 64, 56, 56, 64));
    this->model.push_back(new Conv2D(3, 128, 1, 1, "5", MLH, 56, 56, 128, 56, 56, 64));
    this->model.push_back(new Conv2D(1, 128, 1, 0, "6", MLH, 56, 56, 128, 56, 56, 128));
    this->model.push_back(new Conv2D(3, 256, 1, 1, "7", MLH, 56, 56, 256, 56, 56, 128));
    this->model.push_back(new MaxPool2D(56, 56, 256, 28, 28, 256));
    this->model.push_back(new Conv2D(1, 128, 1, 0, "9", MLH, 28, 28, 128, 28, 28, 256));
    this->model.push_back(new Conv2D(3, 256, 1, 1, "10", MLH, 28, 28, 256, 28, 28, 128));
    this->model.push_back(new Conv2D(1, 256, 1, 0, "11", MLH, 28, 28, 256, 28, 28, 256));
    this->model.push_back(new Conv2D(3, 256, 1, 1, "12", MLH, 28, 28, 256, 28, 28, 256));
    this->model.push_back(new MaxPool2D(28, 28, 256, 14, 14, 256));
    this->model.push_back(new Conv2D(1, 256, 1, 0, "14", MLH, 14, 14, 256, 14, 14, 256));
    this->model.push_back(new Conv2D(3, 256, 1, 1, "15", MLH, 14, 14, 256, 14, 14, 256));
    this->model.push_back(new Conv2D(3, 256, 2, 1, "16", MLH, 7, 7, 256, 14, 14, 256));
    this->model.push_back(new Conv2D(3, 64, 1, 1, "17", MLH, 7, 7, 64, 7, 7, 256));
    // Fully connected layers
    this->model.push_back(new FullyConnected(64 * 7 * 7, 512, MLH, "1", true));
    this->model.push_back(new FullyConnected(512, 7 * 7 * 2 * 5, MLH, "4", false));

    // Load the weights
    for (const auto layer : model)
    {
        layer->loadData();
    }

    // Allocate memory for the output
    this->hostOutput = static_cast<__half *>(malloc(7 * 7 * 10 * sizeof(__half)));
}

yolo::~yolo()
{
    // Clean up layers
    for (auto layer : this->model)
    {
        delete layer;
    }

    if (this->hostOutput)
    {
        free(this->hostOutput);
    }
}

std::vector<std::vector<float>> yolo::getBoxPredictions(__half *inputImage)
{
    __half *output = nullptr;
    for (const auto &layer : this->model)
    {
        output = layer->forward(inputImage);
        inputImage = output;
        hipDeviceSynchronize();
    }

    // Copy the data from GPU to CPU
    hipMemcpy(this->hostOutput, output, 7 * 7 * 10 * sizeof(__half), hipMemcpyDeviceToHost);

    // Convert half to float
    std::vector<std::vector<float>> floatOutput = convertHalfToFloat(aiHelperUtils::reshapeOutput(this->hostOutput, 7 * 7 * 10));

    // Get the final bounding boxes
    return aiHelperUtils::getFinalBoundingBoxes(floatOutput);
}